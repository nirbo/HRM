#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2023 BlinkDL
// SPDX-License-Identifier: MIT
//
// Vendored from https://github.com/BlinkDL/RWKV-CUDA (wkv6_cuda_v1a.cu)
// with only comment header added for integration into HRM.

#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
typedef at::BFloat16 bf16;

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u,
                               F *__restrict__ const _y)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _u += h*_N_;

    __shared__ float r[_N_], k[_N_], u[_N_], w[_N_];
    float state[_N_] = {0};

    __syncthreads();
    u[i] = float(_u[i]);
    __syncthreads();

    for (int t = b*T*C + h*_N_ + i; t < (b+1)*T*C + h*_N_ + i; t += C)
    {
        __syncthreads();
        w[i] = exp(_w[t]);
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float v = float(_v[t]);
        float y = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j+=4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& k_ = (float4&)(k[j]);
            const float4& w_ = (float4&)(w[j]);
            const float4& u_ = (float4&)(u[j]);
            float4& s = (float4&)(state[j]);
            float4 x;

            x.x = k_.x * v;
            x.y = k_.y * v;
            x.z = k_.z * v;
            x.w = k_.w * v;

            y += r_.x * (u_.x * x.x + s.x);
            y += r_.y * (u_.y * x.y + s.y);
            y += r_.z * (u_.z * x.z + s.z);
            y += r_.w * (u_.w * x.w + s.w);

            s.x = s.x * w_.x + x.x;
            s.y = s.y * w_.y + x.y;
            s.z = s.z * w_.z + x.z;
            s.w = s.w * w_.w + x.w;
        }
        _y[t] = F(y);
    }
}

template <typename F>
__global__ void kernel_backward_111(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gr, F *__restrict__ const _gk, F *__restrict__ const _gv, F *__restrict__ const _gu)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _u += h*_N_;

    __shared__ float u_[_N_];
    __shared__ float r[_N_], k[_N_], v[_N_], w_[_N_], gy[_N_];
    __syncthreads();
    u_[i] = float(_u[i]);
    __syncthreads();

    const float u = u_[i];

    float state[_N_] = {0}, scccc[_N_] = {0}, sdddd[_N_] = {0};

    const int t_0 = b*T*C + h*_N_ + i;
    const int t_T_1 = t_0 + (T-1)*C;
    const int t_T = t_0 + T*C;

    float gu = 0;
    for (int t = t_0; t < t_T; t += C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float k = float(_k[t]);
        const float w = exp(_w[t]);
        float gr = 0, gu_ = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = state[j];
            float x = k * v[j];

            gr += (u * x + s) * gy[j];
            gu_ += x * gy[j];
            s = s * w + x;
        }
        _gr[t] = F(gr);
        gu += float(_r[t]) * gu_;
    }
    _gu[b*C + h*_N_ + i] = F(gu);

    for (int t = t_T_1; t >= t_0; t -= C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float rr = float(_r[t]);
        const float w = exp(_w[t]);
        float gk = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = scccc[j];
            float x = rr * gy[j];
            
            gk += (u * x + s) * v[j];
            s = x + s * w;
        }
        _gk[t] = F(gk);
    }

    for (int t = t_T_1; t >= t_0; t -= C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        w_[i] = exp(_w[t]);
        __syncthreads();

        const float gyy = float(_gy[t]);
        float gv = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = sdddd[j];
            float x = gyy * r[j];
            
            gv += (u_[j] * x + s) * k[j];
            s = x + s * w_[j];
        }
        _gv[t] = F(gv);
    }
}

template <typename F>
__global__ void kernel_backward_222(const int B, const int T, const int C, const int H, float *__restrict__ const _buf,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gw)
{
    const int b = blockIdx.x;
    for (int h = 0; h < H; h++) {
        const int i = threadIdx.x;
        float *__restrict__ const buf = _buf + b*(_N_*_T_*_N_) + i*(_T_*_N_);

        __shared__ float v[_N_], gy[_N_];
        float saaaa[_N_] = {0}, sbbbb[_N_] = {0};

        const int t_0 = b*T*C + h*_N_ + i;
        const int t_1 = t_0 + C;
        const int t_2 = t_0 + 2*C;
        const int t_T_2 = t_0 + (T-2)*C;
        const int t_T_1 = t_0 + (T-1)*C;

        for (int t = t_0; t < t_T_2; t += C)
        {
            __syncthreads();
            v[i] = float(_v[t]);
            __syncthreads();

            const float k = float(_k[t]);
            const float w = exp(_w[t]);
            const int tt = (t-t_0)/C*_N_;

            #pragma unroll
            for (int j = 0; j < _N_; j++)
            {
                float& s = saaaa[j];
                float x = k * v[j];
                
                float tmp = w * s + x;
                s = tmp;
                buf[tt + j] = tmp;
                // printf("b %d h %d i %d t %d j %d buf %f\n", b, h, i, tt/_N_, j, tmp);
            }
        }

        for (int t = t_T_1; t > t_1; t -= C)
        {
            __syncthreads();
            gy[i] = float(_gy[t]);
            __syncthreads();

            const float r = float(_r[t]);
            const float w = exp(_w[t]);
            float sum = 0.0f;
            const int tt = (t-t_2)/C*_N_;

            #pragma unroll
            for (int j = 0; j < _N_; j++)
            {
                float& s = sbbbb[j];
                float x = r * gy[j];
                
                float tmp = w * s + x;
                s = tmp;
                sum += buf[tt + j] * tmp;
                // printf("b %d h %d i %d t %d j %d buf %f tmp %f\n", b, h, i, tt/_N_, j, buf[tt + j], tmp);
            }
            _gw[t-C] = F(sum * _w[t-C] * exp(_w[t-C]));
        }
    }
}

void cuda_forward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, float *w, bf16 *u, bf16 *y)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, y);
}

void cuda_backward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, float *w, bf16 *u, bf16 *gy, bf16 *gr, bf16 *gk, bf16 *gv, bf16 *gw, bf16 *gu)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_backward_111<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, gy, gr, gk, gv, gu);

    void* buf = 0;
    hipMalloc(&buf, 4*B*_N_*_T_*_N_);
    kernel_backward_222<<<dim3(B), dim3(_N_)>>>(B, T, C, H, (float *)(buf), r, k, v, w, u, gy, gw);
    hipFree(buf);
}
